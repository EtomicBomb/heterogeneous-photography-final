#include <hip/hip_runtime.h>
#include <stdio.h>
#include <complex>
#include <cmath>
#include <stdio.h>
#include <sched.h>
#include <vector>
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <cmath>

#define I(r, s, d) [(r) * cols_dst * cols_src + (s) * cols_dst + (d)]
#define Isrc(r, s) [(r) * cols_src + (s)]
#define Idst(r, d) [(r) * cols_dst + (d)]

#define CHECK(e) do { hipError_t err; if ((err = e) != hipSuccess) { printf("line %d: %s\n", __LINE__, hipGetErrorString(err)); exit(1);} } while (0)

__device__ inline char 
argmin3(double x, double y, double z) {
    if (x < y) {
        return x < z ? 0 : 2;
    } 
    return y < z ? 1 : 2;
}

__device__ inline void 
min3(double x, double y, double z, double *min, char *argmin) {
    double numbers[] = {x, y, z};
    char index = argmin3(x, y, z);
    *min = numbers[index];
    *argmin = index;
}

// [ ] thread block nonmultiple edge case
// [ ] pixel similarity based on something less aggressive than x^2
// [ ] launch kernels with the right grid / block shape
// [ ] profile to see what bottlenecks are
// [ ] is double range / precision large enough to handle the fact that we are doing the cumulative sum for 2d array? 
// only process some subset of the rows at once to limit our memory usage
// parallelize cumulative sum functions? not sure if that's needed

__global__ void
get_patch_similarity(long rows, long cols_src, long cols_dst, long patch_size, const double *pixel_similarity, double *patch_similarity) {
    long r = blockIdx.y * blockDim.y + threadIdx.y;
    long s = blockIdx.x * blockDim.x + threadIdx.x;
    long d = blockIdx.z * blockDim.z + threadIdx.z;

    if (r >= rows || s >= cols_src || d >= cols_dst) return;

    long rm = r - patch_size - 1;
    long sm = s - patch_size - 1;
    long dm = d - patch_size - 1;

    long overflow = max(0l, max(s + patch_size - cols_src + 1, d + patch_size - cols_dst + 1));
    long underflow = - min(0l, 1 + min(sm, dm));

    long rp = min(r + patch_size, rows - 1);
    long sp = s + patch_size - overflow;
    long dp = d + patch_size - overflow;

    double rpsp = pixel_similarity I(rp, sp, dp);
    double rpsm = sm >= 0 && dm >= 0 ? pixel_similarity I(rp, sm, dm) : 0.0;
    double rmsp = rm >= 0 ? pixel_similarity I(rm, sp, dp) : 0.0;
    double rmsm = rm >= 0 && sm >= 0 && dm >= 0 ? pixel_similarity I(rm, sm, dm) : 0.0;
    double sum = rpsp + rmsm - rpsm - rmsp;

    long width = rp - max(rm, -1l);
    long height = 2 * patch_size + 1 - underflow - overflow;
    long count = width * height;

    patch_similarity I(r, s, d) = sum / count;
}

__global__ void
get_patch_similarity_naive(long rows, long cols_src, long cols_dst, long patch_size, const double *pixel_similarity, double *patch_similarity) {
    long r = blockIdx.y * blockDim.y + threadIdx.y;
    long s = blockIdx.x * blockDim.x + threadIdx.x;
    long d = blockIdx.z * blockDim.z + threadIdx.z;

    if (r >= rows || s >= cols_src || d >= cols_dst) return;

    double sum = 0.0;
    double count = 0.0;
    for (long rn = r - patch_size; rn <= r + patch_size; rn++) {
        for (long sn = s - patch_size; sn <= s + patch_size; sn++) {
            long dn = sn + d - s;
            if (rn >= rows || rn < 0 || sn >= cols_src || sn < 0 || dn < 0 || dn >= cols_dst) continue;
            sum += pixel_similarity I(r, s, d);
            count += 1;
        }
    }

    patch_similarity I(r, s, d) = sum / count;
}

__global__ void
find_costs(long rows, long cols_src, long cols_dst, long patch_size, double occlusion_cost, const double *patch_similarity, char *traceback) {
    long r = blockIdx.y * blockDim.y + threadIdx.y;
    long s = threadIdx.x;

    extern __shared__ double shared_memory[];
    double *prev_prev = &shared_memory[0 * (cols_src + 1)];
    double *prev = &shared_memory[1 * (cols_src + 1)];
    double *current = &shared_memory[2 * (cols_src + 1)];
    if (s == 0) {
        prev_prev[0] = prev[0] = current[0] = INFINITY;
    }
    prev_prev[s + 1] = prev[s + 1] = current[s + 1] = INFINITY;

    for (long k = 0; k < cols_src + cols_dst - 1; k++) {
        long d = k - s;
        __syncthreads();
        if (r < rows && s < cols_src && d < cols_dst && d >= 0) {
            double match = prev_prev[s] + patch_similarity I(r, s, d);
            double occlusion_src = d == 0 ? s * occlusion_cost : prev[s] + occlusion_cost;
            double occlusion_dst = s == 0 ? d * occlusion_cost : prev[s + 1] + occlusion_cost;
            min3(match, occlusion_src, occlusion_dst, &current[s + 1], &traceback I(r, s, d));
        }
        double *new_current = prev_prev;
        prev_prev = prev;
        prev = current;
        current = new_current;
    }
}

__global__ void
traceback_correspondence(long rows, long cols_src, long cols_dst, const char *traceback, long *correspondence, char *valid) {
    long r = blockIdx.y * blockDim.y + threadIdx.y;

    if (r >= rows) return;

    long s = cols_src - 1;
    long d = cols_dst - 1;
    while (s != 0 && d != 0) { // yes
        long direction = traceback I(r, s, d);
        long us[] = {1, 1, 0}; 
        long ud[] = {1, 0, 1};
        s -= us[direction]; 
        d -= ud[direction]; 
        correspondence Isrc(r, s) = d;
        if (direction == 0) {
            valid Isrc(r, s) = 1;
        }
    }
}

__global__ void 
get_pixel_similarity(long rows, long cols_src, long cols_dst, const double *src, const double *dst, double *pixel_similarity) {
    long r = blockIdx.y * blockDim.y + threadIdx.y;
    long s = blockIdx.x * blockDim.x + threadIdx.x;
    long d = blockIdx.z * blockDim.z + threadIdx.z;

    if (r >= rows || s >= cols_src || d >= cols_dst) return;

    double distance = src Isrc(r, s) - dst Idst(r, d);
    pixel_similarity I(r, s, d) = distance * distance;
}

__device__ void 
sum_diagonal(long rows, long cols_src, long cols_dst, long r, long s, long d, double *array) {
    s += 1;
    d += 1;
    while (s < cols_src && d < cols_dst) {
        array I(r, s, d) += array I(r, s - 1, d - 1);
        s += 1;
        d += 1;
    }
}

__global__ void 
cumulative_sum_cols_src(long rows, long cols_src, long cols_dst, double *array) {
    long r = blockIdx.y * blockDim.y + threadIdx.y;
    long s = blockIdx.x * blockDim.x + threadIdx.x;
    long d = 0;
    if (r >= rows || s >= cols_src) return;
    if (s == 0) return; // only want to sum central diagonal once
    sum_diagonal(rows, cols_src, cols_dst, r, s, d, array);
}

__global__ void 
cumulative_sum_cols_dst(long rows, long cols_src, long cols_dst, double *array) {
    long r = blockIdx.y * blockDim.y + threadIdx.y;
    long s = 0;
    long d = blockIdx.z * blockDim.z + threadIdx.z;
    if (r >= rows || d >= cols_dst) return;
    sum_diagonal(rows, cols_src, cols_dst, r, s, d, array);
}

__global__ void 
cumulative_sum_rows(long rows, long cols_src, long cols_dst, double *array) {
    long s = blockIdx.x * blockDim.x + threadIdx.x;
    long d = blockIdx.z * blockDim.z + threadIdx.z;
    if (s >= cols_src || d >= cols_dst) return;
    for (long r = 1; r < rows; r++) {
        array I(r, s, d) += array I(r - 1, s, d);
    }
}

extern "C" int
scanline_stereo_naive(long rows, long cols_src, long cols_dst, long patch_size, double occlusion_cost, const double *src, const double *dst, long *correspondence, char *valid, float *timings) {
    int ncuda_devices = 0;
    CHECK(hipGetDeviceCount(&ncuda_devices));
    if (ncuda_devices == 0) {
        return -1;
    }
    hipSetDevice(0);

    double *src_device;
    CHECK(hipMalloc(&src_device, rows * cols_src * sizeof(*src_device)));
    CHECK(hipMemcpy(src_device, src, rows * cols_src * sizeof(*src_device), hipMemcpyHostToDevice));
    double *dst_device;
    CHECK(hipMalloc(&dst_device, rows * cols_dst * sizeof(*dst_device)));
    CHECK(hipMemcpy(dst_device, dst, rows * cols_dst * sizeof(*dst_device), hipMemcpyHostToDevice));

    double *pixel_similarity;
    CHECK(hipMalloc(&pixel_similarity, rows * cols_src * cols_dst * sizeof(*pixel_similarity)));
    double *patch_similarity;
    CHECK(hipMalloc(&patch_similarity, rows * cols_src * cols_dst * sizeof(*patch_similarity)));
    char *traceback;
    CHECK(hipMalloc(&traceback, rows * cols_src * cols_dst * sizeof(*traceback)));
    long *correspondence_device; 
    CHECK(hipMalloc(&correspondence_device, rows * cols_src * sizeof(*correspondence_device)));
    char *valid_device;
    CHECK(hipMalloc(&valid_device, rows * cols_src * sizeof(*valid_device)));
    CHECK(hipMemset(valid_device, 0, rows * cols_src * sizeof(*valid_device)));

    dim3 block, grid;
    long shared;

    size_t timing_event_count = 8;
    std::vector<hipEvent_t> events(timing_event_count);
    for (hipEvent_t& event : events) {
        hipEventCreate(&event);
    }
    hipEventRecord(events[0]);

    block = dim3(32, 1, 32);
    grid = dim3((cols_src + block.x - 1) / block.x, (rows + block.y - 1) / block.y, (cols_dst + block.z - 1) / block.z);
    get_pixel_similarity<<<grid, block, 0, 0>>>(rows, cols_src, cols_dst, src_device, dst_device, pixel_similarity);

    hipEventRecord(events[1]);

    block = dim3(32, 1, 32);
    grid = dim3((cols_src + block.x - 1) / block.x, (rows + block.y - 1) / block.y, (cols_dst + block.z - 1) / block.z);
    get_patch_similarity_naive<<<grid, block, 0, 0>>>(rows, cols_src, cols_dst, patch_size, pixel_similarity, patch_similarity);

    hipEventRecord(events[2]);
    
    block = dim3(cols_src, 1, 1);
    grid = dim3(1, rows, 1);
    shared = 3 * (cols_src + 1) * sizeof(double);
    find_costs<<<grid, block, shared, 0>>>(rows, cols_src, cols_dst, patch_size, occlusion_cost, patch_similarity, traceback);

    hipEventRecord(events[3]);

    block = dim3(1, 1024, 1);
    grid = dim3(1, (rows + block.y - 1) / block.y, 1);
    traceback_correspondence<<<grid, block, 0, 0>>>(rows, cols_src, cols_dst, traceback, correspondence_device, valid_device);

    hipEventRecord(events[4]);

    CHECK(hipMemcpy(correspondence, correspondence_device, rows * cols_src * sizeof(*correspondence), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(valid, valid_device, rows * cols_src * sizeof(*valid), hipMemcpyDeviceToHost));

    hipFree(src_device);
    hipFree(dst_device);
    hipFree(pixel_similarity);
    hipFree(patch_similarity);
    hipFree(traceback);
    hipFree(correspondence_device);
    hipFree(valid_device);

    for (size_t i = 1; i < timing_event_count; i++) {
        hipEventElapsedTime(&timings[i-1], events[i - 1], events[i]);
    }

    return 0;
}

extern "C" int
scanline_stereo(long rows, long cols_src, long cols_dst, long patch_size, double occlusion_cost, const double *src, const double *dst, long *correspondence, char *valid, float *timings) {
    int ncuda_devices = 0;
    CHECK(hipGetDeviceCount(&ncuda_devices));
    if (ncuda_devices == 0) {
        return -1;
    }
    hipSetDevice(0);

    double *src_device;
    CHECK(hipMalloc(&src_device, rows * cols_src * sizeof(*src_device)));
    CHECK(hipMemcpy(src_device, src, rows * cols_src * sizeof(*src_device), hipMemcpyHostToDevice));
    double *dst_device;
    CHECK(hipMalloc(&dst_device, rows * cols_dst * sizeof(*dst_device)));
    CHECK(hipMemcpy(dst_device, dst, rows * cols_dst * sizeof(*dst_device), hipMemcpyHostToDevice));

    double *pixel_similarity;
    CHECK(hipMalloc(&pixel_similarity, rows * cols_src * cols_dst * sizeof(*pixel_similarity)));
    double *patch_similarity;
    CHECK(hipMalloc(&patch_similarity, rows * cols_src * cols_dst * sizeof(*patch_similarity)));
    char *traceback;
    CHECK(hipMalloc(&traceback, rows * cols_src * cols_dst * sizeof(*traceback)));
    long *correspondence_device; 
    CHECK(hipMalloc(&correspondence_device, rows * cols_src * sizeof(*correspondence_device)));
    char *valid_device;
    CHECK(hipMalloc(&valid_device, rows * cols_src * sizeof(*valid_device)));
    CHECK(hipMemset(valid_device, 0, rows * cols_src * sizeof(*valid_device)));

    dim3 block, grid;
    long shared;

    size_t timing_event_count = 8;
    std::vector<hipEvent_t> events(timing_event_count);
    for (hipEvent_t& event : events) {
        hipEventCreate(&event);
    }
    hipEventRecord(events[0]);

    block = dim3(32, 1, 32);
    grid = dim3((cols_src + block.x - 1) / block.x, (rows + block.y - 1) / block.y, (cols_dst + block.z - 1) / block.z);
    get_pixel_similarity<<<grid, block, 0, 0>>>(rows, cols_src, cols_dst, src_device, dst_device, pixel_similarity);

    hipEventRecord(events[1]);

    block = dim3(1, 32, 32);
    grid = dim3((cols_src + block.x - 1) / block.x, 1, (cols_dst + block.z - 1) / block.z);
    cumulative_sum_rows<<<grid, block, 0, 0>>>(rows, cols_src, cols_dst, pixel_similarity);

    hipEventRecord(events[2]);

    block = dim3(32, 32, 1);
    grid = dim3((cols_src + block.x - 1) / block.x, (rows + block.y - 1) / block.y, 1);
    cumulative_sum_cols_src<<<grid, block, 0, 0>>>(rows, cols_src, cols_dst, pixel_similarity);
    
    hipEventRecord(events[3]);

    block = dim3(1, 32, 32);
    grid = dim3(1, (rows + block.y - 1) / block.y, (cols_dst + block.z - 1) / block.z);
    cumulative_sum_cols_dst<<<grid, block, 0, 0>>>(rows, cols_src, cols_dst, pixel_similarity);

    hipEventRecord(events[4]);

    block = dim3(32, 1, 32);
    grid = dim3((cols_src + block.x - 1) / block.x, (rows + block.y - 1) / block.y, (cols_dst + block.z - 1) / block.z);
    get_patch_similarity<<<grid, block, 0, 0>>>(rows, cols_src, cols_dst, patch_size, pixel_similarity, patch_similarity);

    hipEventRecord(events[5]);
    
    block = dim3(cols_src, 1, 1);
    grid = dim3(1, rows, 1);
    shared = 3 * (cols_src + 1) * sizeof(double);
    find_costs<<<grid, block, shared, 0>>>(rows, cols_src, cols_dst, patch_size, occlusion_cost, patch_similarity, traceback);

    hipEventRecord(events[6]);

    block = dim3(1, 1024, 1);
    grid = dim3(1, (rows + block.y - 1) / block.y, 1);
    traceback_correspondence<<<grid, block, 0, 0>>>(rows, cols_src, cols_dst, traceback, correspondence_device, valid_device);

    hipEventRecord(events[7]);

    CHECK(hipMemcpy(correspondence, correspondence_device, rows * cols_src * sizeof(*correspondence), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(valid, valid_device, rows * cols_src * sizeof(*valid), hipMemcpyDeviceToHost));

    hipFree(src_device);
    hipFree(dst_device);
    hipFree(pixel_similarity);
    hipFree(patch_similarity);
    hipFree(traceback);
    hipFree(correspondence_device);
    hipFree(valid_device);

    for (size_t i = 1; i < timing_event_count; i++) {
        hipEventElapsedTime(&timings[i-1], events[i - 1], events[i]);
    }

    return 0;
}
